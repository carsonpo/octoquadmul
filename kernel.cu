#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cuda/pipeline>
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <cmath>
#include <chrono>
#include "helpers.cu"
#include <iostream>
#include <vector>
#include <cstdint>
#include "configs.cu"
#include "hipblas.h"

#define div_ru(a, b) (((a) + (b) - 1) / (b))

#define WARP_SIZE 32
#define DEBUG false

// M is not constexpr-d because tokens * batch can vary, but the rest of the problem size is fixed for specific configs
template <int BlockRowWarps, int BlockColWarps, int WarpRowTiles, int WarpColTiles, int PatchM, int PatchN, int ChunkK, int NumStages, int PipelineStrategy, int kWMMA_M, int kWMMA_N, int kWMMA_K, int kN, int kK>
struct IGemmConfig
{
    static constexpr int kBlockRowWarps = BlockRowWarps;
    static constexpr int kBlockColWarps = BlockColWarps;
    static constexpr int kWarpRowTiles = WarpRowTiles;
    static constexpr int kWarpColTiles = WarpColTiles;
    static constexpr int kChunkK = ChunkK;
    static constexpr int kNumStages = NumStages;
    static constexpr int kPipelineStrategy = PipelineStrategy;

    static constexpr int kPatchM = PatchM;
    static constexpr int kPatchN = PatchN;

    // Derived constants
    static constexpr int kBlockRowTiles = kWarpRowTiles * kBlockRowWarps;
    static constexpr int kBlockColTiles = kWarpColTiles * kBlockColWarps;

    static constexpr int kTileSizeM = kWMMA_M * kBlockRowTiles;
    static constexpr int kTileSizeN = kWMMA_N * kBlockColTiles;
    static constexpr int kTileSizeK = kWMMA_K * kChunkK;

    static constexpr int K = kK;
    static constexpr int N = kN;
    static constexpr int WMMA_M = kWMMA_M;
    static constexpr int WMMA_N = kWMMA_N;
    static constexpr int WMMA_K = kWMMA_K;
};

// 128-bit vector type for efficient memory loads
using Data128B = int4;
using Data64B = int2;
constexpr int ALIGN_SIZE_A = 16;
constexpr int ALIGN_SIZE_B = 32;
#define PRESHUFFLE false

template <typename Config>
__global__ void igemm(const int8_t *A, const uint8_t *B, int32_t *C, int M)
{
    extern __shared__ int8_t shared_memory[];

    using FragA = nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, Config::WMMA_M, Config::WMMA_N, Config::WMMA_K, int8_t, nvcuda::wmma::row_major>;
    using FragB = nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, Config::WMMA_M, Config::WMMA_N, Config::WMMA_K, int8_t, nvcuda::wmma::col_major>;
    using FragC = nvcuda::wmma::fragment<nvcuda::wmma::accumulator, Config::WMMA_M, Config::WMMA_N, Config::WMMA_K, int32_t>;

    // Set up shared memory tensors for A and B with multiple stages
    SmemTensor3D<int8_t, Config::kNumStages, Config::kTileSizeM, Config::kTileSizeK>
        smemA(shared_memory);
    SmemTensor3D<uint8_t, Config::kNumStages, Config::kTileSizeN, Config::kTileSizeK / 2> smemB(smemA.endPtr);

    // Set up global memory tensors for A, B, and C
    GMemTensor2D<int8_t> gmemA((int8_t *)A, M, Config::K);
    GMemTensor2D<uint8_t> gmemB((uint8_t *)B, Config::N, Config::K / 2); // Note: B is transposed and bit packed int4
    GMemTensor2D<int32_t> gmemC(C, M, Config::N);

    // Calculate warp and lane IDs
    int warp_id = threadIdx.x / WARP_SIZE;
    // int warp_row = warp_id / Config::kBlockColWarps;
    // int warp_col = warp_id % Config::kBlockColWarps;

    int warp_row = warp_id / (Config::kBlockColWarps / Config::kPatchN);
    int warp_col = warp_id % (Config::kBlockColWarps / Config::kPatchN);

    // Calculate starting positions for this block
    int block_row_start = blockIdx.x * Config::kTileSizeM;
    int block_col_start = blockIdx.y * Config::kTileSizeN;

    FragA a_frag[Config::kPatchM][Config::kWarpRowTiles];
    FragB b_frag[Config::kPatchN][Config::kWarpColTiles];
    FragC c_frag[Config::kPatchM][Config::kPatchN][Config::kWarpRowTiles][Config::kWarpColTiles];

    for (int pm = 0; pm < Config::kPatchM; pm++)
    {

        for (int pn = 0; pn < Config::kPatchN; pn++)
        {

            for (int i = 0; i < Config::kWarpRowTiles; i++)
            {

                for (int j = 0; j < Config::kWarpColTiles; j++)
                {
                    nvcuda::wmma::fill_fragment(c_frag[pm][pn][i][j], 0);
                }
            }
        }
    }

    auto load_A_tile = [&](int stage, int k_offset)
    {
        for (int i = threadIdx.x; i < (Config::kTileSizeM * Config::kTileSizeK) / ALIGN_SIZE_A; i += blockDim.x)
        {
            int row = (i * ALIGN_SIZE_A) / Config::kTileSizeK;
            int col = (i * ALIGN_SIZE_A) % Config::kTileSizeK;
            if (block_row_start + row < M && k_offset + col + ALIGN_SIZE_A - 1 < Config::K)
            {
                int8_t *shared_ptr = smemA.get_ptr(stage, row, col);
                int8_t *global_ptr = gmemA.get_ptr(block_row_start + row, k_offset + col);
                __pipeline_memcpy_async(shared_ptr, global_ptr, ALIGN_SIZE_A);
            }
        }
    };

    auto load_B_tile = [&](int stage, int k_offset)
    {
        for (int i = threadIdx.x; i < (Config::kTileSizeN * Config::kTileSizeK) / ALIGN_SIZE_B; i += blockDim.x)
        {
            int row = (i * ALIGN_SIZE_B) / Config::kTileSizeK;
            int col = (i * ALIGN_SIZE_B) % Config::kTileSizeK;
            int global_row = block_col_start + row;
            int global_col = k_offset + col;

            if (global_row < Config::N && global_col < Config::K)
            {
                uint8_t *shared_ptr = smemB.get_ptr(stage, row, col / 2);
                uint8_t *global_ptr = gmemB.get_ptr(global_row, global_col / 2);
                __pipeline_memcpy_async(shared_ptr, global_ptr, ALIGN_SIZE_B / 2);
            }
        }
    };

    auto store_C_tile = [&]()
    {
        for (int pm = 0; pm < Config::kPatchM; pm++)
        {

            for (int pn = 0; pn < Config::kPatchN; pn++)
            {

                for (int i = 0; i < Config::kWarpRowTiles; i++)
                {

                    for (int j = 0; j < Config::kWarpColTiles; j++)
                    {
                        int row = block_row_start + ((warp_row * Config::kPatchM + pm) * Config::kWarpRowTiles + i) * Config::WMMA_M;
                        int col = block_col_start + ((warp_col * Config::kPatchN + pn) * Config::kWarpColTiles + j) * Config::WMMA_N;

                        if (row < M && col < Config::N)
                        {
                            nvcuda::wmma::store_matrix_sync(
                                gmemC.get_ptr(row, col),
                                c_frag[pm][pn][i][j],
                                Config::N,
                                nvcuda::wmma::mem_row_major);
                        }
                    }
                }
            }
        }
        __syncthreads();
    };

    auto unpack_and_load_frag = [&](int stage, int warp_col, int pn, int j, int kk)
    {
        const uint8_t *packed_ptr = smemB.get_ptr(stage,
                                                  warp_col * Config::kWarpColTiles * Config::WMMA_N + j * Config::WMMA_N,
                                                  kk / 2);
        int lane_id = threadIdx.x % WARP_SIZE;
        constexpr int numel = b_frag[pn][j].num_elements / 2;

        constexpr size_t tile_size = Config::WMMA_K * Config::WMMA_N / 2 * Config::kChunkK;

        // I don't know why this works, but it does.
        constexpr int factor = ((Config::kChunkK - 1) * 16);
        const int shift = (lane_id / 4) * factor;

        const int base1 = lane_id * numel + shift;
        const int base2 = tile_size + base1;

        for (int packed_idx = base1; packed_idx < base1 + numel; packed_idx++)
        {
            uint8_t packed = packed_ptr[packed_idx / 2];
            if (packed_idx % 2 == 0)
            {
                b_frag[pn][j].x[packed_idx - base1] = (packed & 0x0F) - 8;
            }
            else
            {
                b_frag[pn][j].x[packed_idx - base1] = (packed >> 4) - 8;
            }
        }
        for (int packed_idx = base2; packed_idx < base2 + numel; packed_idx++)
        {
            uint8_t packed = packed_ptr[packed_idx / 2];
            if (packed_idx % 2 == 0)
            {
                b_frag[pn][j].x[packed_idx - base2 + numel] = (packed & 0x0F) - 8;
            }
            else
            {
                b_frag[pn][j].x[packed_idx - base2 + numel] = (packed >> 4) - 8;
            }
        }
    };

    auto pipeline_strategy_1 = [&]()
    {
        load_A_tile(0, 0);
        load_B_tile(0, 0);
        __pipeline_commit();
        __pipeline_wait_prior(0);
        __syncthreads();

        int current_stage = 0;
        for (int k = 0; k < Config::K; k += Config::kTileSizeK)
        {
            // Start loading next stage if available
            if (k + Config::kTileSizeK < Config::K)
            {
                int next_stage = 1 - current_stage;
                load_A_tile(next_stage, k + Config::kTileSizeK);
                load_B_tile(next_stage, k + Config::kTileSizeK);
                __pipeline_commit();
            }

            for (int kk = 0; kk < Config::kTileSizeK; kk += Config::WMMA_K)
            {

                for (int pm = 0; pm < Config::kPatchM; pm++)
                {

                    for (int i = 0; i < Config::kWarpRowTiles; i++)
                    {
                        nvcuda::wmma::load_matrix_sync(
                            a_frag[pm][i],
                            smemA.get_ptr(current_stage, (warp_row * Config::kPatchM + pm) * Config::kWarpRowTiles * Config::WMMA_M + i * Config::WMMA_M, kk),
                            Config::kTileSizeK);
                    }
                }

                for (int pn = 0; pn < Config::kPatchN; pn++)
                {

                    for (int j = 0; j < Config::kWarpColTiles; j++)
                    {
                        unpack_and_load_frag(current_stage, warp_col * Config::kPatchN + pn, pn, j, kk);
                    }
                }

                for (int pm = 0; pm < Config::kPatchM; pm++)
                {

                    for (int pn = 0; pn < Config::kPatchN; pn++)
                    {

                        for (int i = 0; i < Config::kWarpRowTiles; i++)
                        {

                            for (int j = 0; j < Config::kWarpColTiles; j++)
                            {
                                nvcuda::wmma::mma_sync(c_frag[pm][pn][i][j], a_frag[pm][i], b_frag[pn][j], c_frag[pm][pn][i][j]);
                            }
                        }
                    }
                }
            }

            // Wait for next stage to finish loading
            __pipeline_wait_prior(0);
            __syncthreads();

            // Swap stages
            current_stage = 1 - current_stage;
        }
    };

    pipeline_strategy_1();

    store_C_tile();
}

template <typename Config>
void launch_igemm(const int8_t *A, const uint8_t *B, int32_t *C, int M, hipStream_t stream)
{
    dim3 grid_dim(div_ru(M, Config::kTileSizeM), div_ru(Config::N, Config::kTileSizeN));
    dim3 block_dim(WARP_SIZE * (Config::kBlockRowWarps / Config::kPatchM) * (Config::kBlockColWarps / Config::kPatchN));

    size_t shared_mem_size = Config::kNumStages * (Config::kTileSizeM * Config::kTileSizeK * sizeof(int8_t) + Config::kTileSizeN * Config::kTileSizeK * sizeof(int8_t) / 2);

    igemm<Config><<<grid_dim, block_dim, shared_mem_size, stream>>>(A, B, C, M);
}

#define LAUNCH_KERNEL_IF_CONDITION(config, mCond, nCond, kCond)                        \
    else if (n == nCond && m == mCond && k == kCond)                                   \
    {                                                                                  \
        using ThisConfig = IGemmConfig<config.BlockRowWarps, config.BlockColWarps,     \
                                       config.WarpRowTiles, config.WarpColTiles,       \
                                       config.PatchM, config.PatchN, config.ChunkK,    \
                                       config.NumStages, config.PipelineStrategy,      \
                                       config.kWMMA_M, config.kWMMA_N, config.kWMMA_K, \
                                       config.N, config.K>;                            \
        launch_igemm<ThisConfig>(A_ptr, B_ptr, C_ptr, m, stream);                      \
        return;                                                                        \
    }

void wrapper(void *A, void *B, void *C, const int m, const int n, const int k, hipStream_t stream)
{
    const int8_t *A_ptr = reinterpret_cast<const int8_t *>(A);
    const uint8_t *B_ptr = reinterpret_cast<const uint8_t *>(B);
    int32_t *C_ptr = reinterpret_cast<int32_t *>(C);

    if (false)
    {
    }
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_6144_4096, 64, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_8192_8192, 64, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_10240_8192, 64, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_28672_4096, 64, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_4096_14336, 64, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_8192_28672, 64, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_64_57344_8192, 64, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_6144_4096, 128, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_8192_8192, 128, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_10240_8192, 128, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_28672_4096, 128, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_4096_14336, 128, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_8192_28672, 128, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_128_57344_8192, 128, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_6144_4096, 256, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_8192_8192, 256, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_10240_8192, 256, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_28672_4096, 256, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_4096_14336, 256, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_8192_28672, 256, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_256_57344_8192, 256, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_6144_4096, 512, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_8192_8192, 512, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_10240_8192, 512, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_28672_4096, 512, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_4096_14336, 512, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_8192_28672, 512, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_512_57344_8192, 512, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_6144_4096, 1024, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_8192_8192, 1024, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_10240_8192, 1024, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_28672_4096, 1024, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_4096_14336, 1024, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_8192_28672, 1024, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_1024_57344_8192, 1024, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_6144_4096, 2048, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_8192_8192, 2048, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_10240_8192, 2048, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_28672_4096, 2048, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_4096_14336, 2048, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_8192_28672, 2048, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_2048_57344_8192, 2048, 57344, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_6144_4096, 4096, 6144, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_8192_8192, 4096, 8192, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_10240_8192, 4096, 10240, 8192)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_28672_4096, 4096, 28672, 4096)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_4096_14336, 4096, 4096, 14336)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_8192_28672, 4096, 8192, 28672)
    LAUNCH_KERNEL_IF_CONDITION(octomul_4096_57344_8192, 4096, 57344, 8192)
}

hipblasHandle_t g_cublas_handle = nullptr;

void init_cublas()
{
    if (g_cublas_handle == nullptr)
    {
        hipblasStatus_t status = hipblasCreate(&g_cublas_handle);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("cuBLAS initialization failed with error code %d\n", status);
        }
    }
}

void destroy_cublas()
{
    if (g_cublas_handle != nullptr)
    {
        hipblasDestroy(g_cublas_handle);
        g_cublas_handle = nullptr;
    }
}

void cublas_igemm(const int8_t *A, const int8_t *B, int32_t *C, int M, int N, int K, hipStream_t stream)
{
    if (g_cublas_handle == nullptr)
    {
        printf("cuBLAS handle not initialized\n");
        return;
    }

    hipblasStatus_t status = hipblasSetStream(g_cublas_handle, stream);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cuBLAS set stream failed with error code %d\n", status);
        return;
    }

    const int32_t alpha = 1;
    const int32_t beta = 0;

    status = hipblasGemmEx(g_cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                          N, M, K,
                          &alpha,
                          B, HIP_R_8I, K,
                          A, HIP_R_8I, K,
                          &beta,
                          C, HIP_R_32I, N,
                          HIP_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cuBLAS GEMM failed with error code %d\n", status);
    }
}
