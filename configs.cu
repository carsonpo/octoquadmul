
#include <hip/hip_runtime.h>
struct KernelConfig
{
    const int BlockRowWarps;
    const int BlockColWarps;
    const int WarpRowTiles;
    const int WarpColTiles;
    const int PatchM;
    const int PatchN;
    const int ChunkK;
    const int NumStages;
    const int PipelineStrategy;
    const int kWMMA_M;
    const int kWMMA_N;
    const int kWMMA_K;
    const int K;
    const int N;
};

constexpr KernelConfig octomul_4096_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 2,
    /* ChunkK */ 4,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_4096_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_4096_28672_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_4096_10240_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 6,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_4096_6144_4096 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_2048_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_2048_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_2048_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 6,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_2048_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 6,
    /* WarpColTiles */ 4,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_1024_8192_28672 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_1024_8192_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_128_10240_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_128_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 2,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_512_8192_28672 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_4096_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_128_28672_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_1024_10240_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_64_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 4,
    /* PatchM */ 1,
    /* PatchN */ 2,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_256_10240_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_1024_4096_14336 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_1024_28672_4096 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_256_57344_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_64_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_256_28672_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_128_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_64_28672_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_512_6144_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_2048_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 6,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_64_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_256_6144_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_64_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_64_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_128_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_128_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_1024_6144_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_64_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_256_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_2048_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_1024_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_256_8192_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_2048_4096_14336 = {
    /* BlockRowWarps */ 6,
    /* BlockColWarps */ 6,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_512_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 6,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_128_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_256_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_512_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 6,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_4096_8192_28672 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 6,
    /* WarpColTiles */ 2,
    /* PatchM */ 1,
    /* PatchN */ 2,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_512_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_512_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 2,
    /* PatchM */ 2,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_512_10240_8192 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 2,
    /* WarpColTiles */ 2,
    /* PatchM */ 4,
    /* PatchN */ 1,
    /* ChunkK */ 2,
    /* NumStages */ 4,
    /* PipelineStrategy */ 1,
    /* kWMMA_M */ 16,
    /* kWMMA_N */ 16,
    /* kWMMA_K */ 16,
    /* K */ 8192,
    /* N */ 10240};
